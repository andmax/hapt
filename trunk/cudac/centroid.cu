#include "hip/hip_runtime.h"
/**
 *   HAPT -- Centroid CUDA
 *
 * Maximo, Andre -- Mar, 2009
 *
 */

#include <hip/hip_runtime.h>
#include <cutil.h>

#include <stdio.h>
#include <sys/time.h>

#include "centroid.cuh"

typedef unsigned long long int uint_64;

#include "bitonic_sort.cu"

#include "quick_sort.cu"

#define BLOCK_X 16
#define BLOCK_Y 16
#define NTHREADS BLOCK_X*BLOCK_Y

uint3 dimBlock = { BLOCK_X, BLOCK_Y, 1 };
uint3 dimGrid = { 1, 1, 1 };

uint numCentroids;
uint szCentroidList, szUnpackedArray;

float *d_centroidList;

uint *d_unpackedArray;

texture<float4, 1, hipReadModeElementType> centroidTex;

//Round a / b to nearest higher integer value
__host__ inline uint
iDivUp(uint a, uint b) { return (a % b != 0) ? (a / b + 1) : (a / b); }

// Align a to nearest higher multiple of b
__host__ inline uint
iAlignUp(uint a, uint b) { return (a % b != 0) ? (a - a % b + b) : a; }

/// Update centroid values inside the GPU

__global__
void updateCentroid( uint_64 *packedArray,
		     const float mvX, const float mvY, const float mvZ,
		     const uint nC ) {

	uint centroidId = __mul24( __mul24( __mul24( blockIdx.y, gridDim.x ) + blockIdx.x, blockDim.x )
				   + threadIdx.y, blockDim.y ) + threadIdx.x;

	if( centroidId >= nC ) centroidId = nC - 1;

 	float4 centroid = tex1Dfetch(centroidTex, centroidId );

	float cZ = mvX * centroid.x + mvY * centroid.y + mvZ * centroid.z;

	if( cZ < -1.0 ) cZ = -1.0;
	if( cZ >  1.0 ) cZ =  1.0;

	uint_64 pack = (uint_64)( ((cZ + 1.0) / 2.0) * 0xFFFFFFFF );
	pack <<= 32;
	pack += (uint_64) centroidId;

	packedArray[ centroidId ] = pack;

}

/// Unpack array packed in update step inside the GPU

__global__
void unpackArray( uint *unpackedArray,
		  const uint_64 *packedArray,
		  const uint nC ) {

	uint centroidId = __mul24( __mul24( __mul24( blockIdx.y, gridDim.x ) + blockIdx.x, blockDim.x )
				   + threadIdx.y, blockDim.y ) + threadIdx.x;

	if( centroidId >= nC ) centroidId = nC - 1;

	/// Just throw away centroidZ and get sorted centroidId from packedArray
	unpackedArray[ centroidId ] = (uint) ( packedArray[ centroidId ] );

}

/// Initialize General

extern "C"
__host__
void initCUDA( float *h_centroidList, uint _numCentroids ) {

	/// General
	numCentroids = _numCentroids;

	szCentroidList = numCentroids * 4 * sizeof(float);

	szUnpackedArray = numCentroids * sizeof(uint);

	dimGrid.x = iDivUp( numCentroids, NTHREADS );

	CUDA_SAFE_CALL( hipMalloc((void**) &d_centroidList, szCentroidList) );
 	CUDA_SAFE_CALL( hipMemcpy(d_centroidList, h_centroidList, szCentroidList, hipMemcpyHostToDevice) );

	CUDA_SAFE_CALL( hipMalloc((void**) &d_unpackedArray, szUnpackedArray) );

	initBitonic( numCentroids );

	initQuick( numCentroids );

}

/// Run bitonic sort using CUDA

extern "C"
__host__
void bitonicSortCUDA( uint *ids, float _mvX, float _mvY, float _mvZ ) {

	CUDA_SAFE_CALL( hipBindTexture(0, centroidTex, d_centroidList, szCentroidList) );

	updateCentroid<<< dimGrid, dimBlock >>>( d_packedArrayBitonic, _mvX, _mvY, _mvZ, numCentroids );

	CUDA_SAFE_CALL( hipUnbindTexture(centroidTex) );

	bitonicSort();

	unpackArray<<< dimGrid, dimBlock >>>( d_unpackedArray, d_packedArrayBitonic, numCentroids );

 	CUDA_SAFE_CALL( hipMemcpy(ids, d_unpackedArray, szUnpackedArray, hipMemcpyDeviceToHost) );

}

/// Run quick sort using CUDA

extern "C"
__host__
void quickSortCUDA( uint *ids, float _mvX, float _mvY, float _mvZ ) {

	CUDA_SAFE_CALL( hipBindTexture(0, centroidTex, d_centroidList, szCentroidList) );

	// create and start timer
// 	unsigned timer;
// 	cutCreateTimer(&timer);
// 	cutStartTimer(timer);

	updateCentroid<<< dimGrid, dimBlock >>>( d_packedArrayQuick, _mvX, _mvY, _mvZ, numCentroids );

	CUDA_SAFE_CALL( hipUnbindTexture(centroidTex) );

	quickSort();

	unpackArray<<< dimGrid, dimBlock >>>( d_unpackedArray, d_packedArrayQuick, numCentroids );

	// stop and destroy timer
//	cutStopTimer(timer);
//	printf("quicksort kernel time %f ms\n ", cutGetTimerValue(timer));
//	cutDeleteTimer(timer);

 	CUDA_SAFE_CALL( hipMemcpy(ids, d_unpackedArray, szUnpackedArray, hipMemcpyDeviceToHost) );

}

/// Clean CUDA memory

extern "C"
__host__
void cleanCUDA( void ) {

	CUDA_SAFE_CALL( hipFree(d_centroidList) );
 	CUDA_SAFE_CALL( hipFree(d_unpackedArray) );

	cleanBitonic();

	cleanQuick();

}
