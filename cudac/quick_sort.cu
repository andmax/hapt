#include "hip/hip_runtime.h"
/**
 *   Quick Sort on GPU (using CUDA)
 *
 *   Adapted from:
 *
 *     http://www.cs.chalmers.se/~dcs/gpuqsortdcs.html
 *
 * Maximo, Andre -- Mar, 2009
 *
 **/

#include <algorithm>

#include <cutil.h>

uint szPackedArrayQuick;

uint_64 *d_packedArrayQuick, *d_auxiliaryArray;

#define MAXTHREADS 256
#define MAXBLOCKS 2048

uint threads, blockscount, sbsize, size;

// Keep tracks of the data blocks in phase one
typedef struct _BlockSize {

	unsigned int beg;
	unsigned int end;
	unsigned int orgbeg;
	unsigned int orgend;
	uint_64		 rmaxpiv;
	uint_64		 lmaxpiv;
	uint_64		 rminpiv;
	uint_64		 lminpiv;
	bool		 altered;
	bool		 flip;
	uint_64		 pivot;

} BlockSize;

// Holds parameters to the kernel in phase one
typedef struct _Params {

	unsigned int from;
	unsigned int end;
	uint_64 pivot;
	unsigned int ptr;
	bool last;

} Params;

// Used to perform a cumulative sum between blocks.
// Unnecessary for cards with atomic operations.
// Will be removed when these becomes more common
typedef struct _Length {

	uint_64 maxpiv[MAXBLOCKS];
	uint_64 minpiv[MAXBLOCKS];
	unsigned int left[MAXBLOCKS];
	unsigned int right[MAXBLOCKS];

} Length;

// Since we have divided up the kernel in to three
// we need to remember the result of the cumulative sum
// Unnecessary for cards with atomic operations.
// Will be removed when these becomes more common
typedef struct _Hist {

	unsigned int left[(MAXTHREADS)*MAXBLOCKS];
	unsigned int right[(MAXTHREADS)*MAXBLOCKS];

} Hist;

typedef struct _LQSortParams {

	unsigned int beg;
	unsigned int end;
	bool flip;
	unsigned int sbsize;

} LQSortParams;

#include "gpuqsort_kernels.cu"

#undef THREADS
#define THREADS threads

Params* params;
Params* dparams;
LQSortParams* lqparams;
LQSortParams* dlqparams;
Hist* dhists;
Length* dlength;
Length* length;
BlockSize* workset;
float TK,TM,MK,MM,SM,SK;

/// Initialize Quick

__host__
void initQuick( uint numElements ) {

	size = numElements;

	/// Quick Sort
	szPackedArrayQuick = size * sizeof(uint_64);

	CUDA_SAFE_CALL( hipMalloc((void**) &d_packedArrayQuick, szPackedArrayQuick) );

	/// Prepare device (qsort)
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	if( !strcmp(deviceProp.name, "GeForce 8800 GTX") ) {

		TK = 1.17125033316e-005f;
		TM = 52.855721393f;
		MK = 3.7480010661e-005f;
		MM = 476.338308458f;
		SK = 4.68500133262e-005f;
		SM = 211.422885572f;

	} else if( !strcmp(deviceProp.name, "GeForce 8600 GTS") ) {

		TK = 0.0f;
		TM = 64.0f;
		MK = 0.0000951623403898f;
		MM = 476.338308458f;
		SK = 0.0000321583081317f;
		SM = 202.666666667f;

	} else {

		TK = 0;
		TM = 128;
		MK = 0;
		MM = 512;
		SK = 0;
		SM = 512;

	}

	if( hipHostMalloc((void**)&workset,MAXBLOCKS*2*sizeof(BlockSize)) != hipSuccess ) return;
	if( hipHostMalloc((void**)&params,MAXBLOCKS*sizeof(Params)) != hipSuccess ) return;
	if( hipHostMalloc((void**)&length,sizeof(Length)) != hipSuccess ) return;
	if( hipHostMalloc((void**)&lqparams,MAXBLOCKS*sizeof(LQSortParams)) != hipSuccess ) return;
	if( hipMalloc((void**)&dlqparams,MAXBLOCKS*sizeof(LQSortParams)) != hipSuccess ) return;
	if( hipMalloc((void**)&dhists,sizeof(Hist)) != hipSuccess ) return;
	if( hipMalloc((void**)&dlength,sizeof(Length)) != hipSuccess ) return;
	if( hipMalloc((void**)&dparams,MAXBLOCKS*sizeof(Params)) != hipSuccess ) return;

	threads = 1<<(int)round(log(size * TK + TM)/log(2.0));
	blockscount = 1<<(int)round(log(size * MK + MM)/log(2.0));
	sbsize = 1<<(int)round(log(size * SK + SM)/log(2.0));

	if( threads > MAXTHREADS ) return; 
	if( blockscount > MAXBLOCKS ) return;

	if( hipMalloc((void**)&d_auxiliaryArray, szPackedArrayQuick) != hipSuccess ) return;

}

/// Quick Sort

__host__
void quickSort( void ) {

#ifdef HASATOMICS
	uint_64* doh;
	uint_64 oh;

	hipGetSymbolAddress( (void**)&doh, "ohtotal" );
	oh=0;
	hipMemcpy( doh, &oh, 4, hipMemcpyHostToDevice );
#endif

	//d_auxiliaryArray = 0;
	//if( hipMalloc((void**)&d_auxiliaryArray, szPackedArrayQuick) != hipSuccess ) return;

	hipDeviceSynchronize();

	// We start with a set containg only the sequence to be sorted
	// This will grow as we partition the data
	workset[0].beg = 0;
	workset[0].end = size;
	workset[0].orgbeg = 0;
	workset[0].orgend = size;
	workset[0].altered = false;
	workset[0].flip = false;

	// Get a starting pivot
	workset[0].pivot = 0x7fffffffffffffff;

	unsigned int worksize = 1;

	unsigned int blocks = blockscount/2;
	unsigned totsize = size;
	unsigned int maxlength = (size/blocks)/4;

	unsigned int iterations = 0;
	bool flip = true;

	// Partition the sequences until we have enough
	while( worksize < blocks ) {

		unsigned int ws = totsize / blocks;
		unsigned int paramsize = 0;

		// Go through the sequences we have and divide them into sections
		// and assign thread blocks according to their size
		for(unsigned int i = 0; i < worksize; i++) {

			if( (workset[i].end-workset[i].beg) < maxlength) continue;

			// Larger sequences gets more thread blocks assigned to them
			unsigned int blocksassigned = max( (workset[i].end-workset[i].beg) / ws, 1 );
			for(unsigned int q = 0; q < blocksassigned; q++) {

				params[paramsize].from = workset[i].beg + ws*q;
				params[paramsize].end = params[paramsize].from + ws;
				params[paramsize].pivot = workset[i].pivot;
				params[paramsize].ptr = i;
				params[paramsize].last = false;
				paramsize++;
				
			}

			params[paramsize-1].last = true;
			params[paramsize-1].end = workset[i].end;

			workset[i].lmaxpiv=0;
			workset[i].lminpiv=0xffffffffffffffff;
			workset[i].rmaxpiv=0;
			workset[i].rminpiv=0xffffffffffffffff;
		}

		if( paramsize == 0 ) break;

		// Copy the block assignment to the GPU
		if( hipMemcpy(dparams, params, paramsize*sizeof(Params), hipMemcpyHostToDevice) != hipSuccess ) return;

		// Do the cumulative sum
		if( flip ) part1<<< paramsize, THREADS, 2*( (THREADS+1)*2*4+THREADS*2*4 ) >>>(d_packedArrayQuick, dparams, dhists, dlength);
		else part1<<< paramsize, THREADS, 2*( (THREADS+1)*2*4+THREADS*2*4 ) >>>(d_auxiliaryArray, dparams, dhists, dlength);

		if( hipMemcpy(length, dlength, sizeof(Length), hipMemcpyDeviceToHost) != hipSuccess ) return;

		// Do the block cumulative sum. Done on the CPU since not all cards have support for atomic operations yet. 
		for(unsigned int i = 0; i < paramsize; i++) {

			unsigned int l = length->left[i];
			unsigned int r = length->right[i];
			
			length->left[i] = workset[params[i].ptr].beg;
			length->right[i] = workset[params[i].ptr].end;
			
			workset[params[i].ptr].beg+=l;
			workset[params[i].ptr].end-=r;
			workset[params[i].ptr].altered = true;

			workset[params[i].ptr].rmaxpiv = max(length->maxpiv[i],workset[params[i].ptr].rmaxpiv);
			workset[params[i].ptr].lminpiv = min(length->minpiv[i],workset[params[i].ptr].lminpiv);

			workset[params[i].ptr].lmaxpiv = min(workset[params[i].ptr].pivot,workset[params[i].ptr].rmaxpiv); 
			workset[params[i].ptr].rminpiv = max(workset[params[i].ptr].pivot,workset[params[i].ptr].lminpiv); 

		}

		// Copy the result of the block cumulative sum to the GPU
		if( hipMemcpy(dlength, length, sizeof(Length), hipMemcpyHostToDevice) != hipSuccess ) return;

		// Move the elements to their correct position
		if( flip ) part2<<< paramsize, THREADS >>>(d_packedArrayQuick, d_auxiliaryArray, dparams, dhists, dlength);
		else part2<<< paramsize, THREADS >>>(d_auxiliaryArray, d_packedArrayQuick, dparams, dhists, dlength);

		// Fill in the pivot value between the left and right blocks
		part3<<< paramsize, THREADS >>>(d_packedArrayQuick, dparams, dhists, dlength);

		flip = !flip;

		// Add the sequences resulting from the partitioning to set
		unsigned int oldworksize = worksize;
		totsize = 0;
		for(unsigned int i = 0; i < oldworksize; i++) {

			if( workset[i].altered ) {

				if(workset[i].beg-workset[i].orgbeg>=maxlength)
					totsize += workset[i].beg-workset[i].orgbeg;
				if(workset[i].orgend-workset[i].end>=maxlength)
					totsize += workset[i].orgend-workset[i].end;

				workset[worksize].beg = workset[worksize].orgbeg = workset[i].orgbeg;
				workset[worksize].end = workset[worksize].orgend = workset[i].beg;
				workset[worksize].flip=flip;
				workset[worksize].altered = false;
				workset[worksize].pivot = (workset[i].lminpiv/2+workset[i].lmaxpiv/2);

				worksize++;

				workset[i].orgbeg = workset[i].beg = workset[i].end;
				workset[i].end = workset[i].orgend;
				workset[i].flip = flip;
				workset[i].pivot = (workset[i].rminpiv/2+workset[i].rmaxpiv/2);
				workset[i].altered = false;

			}
		}

		iterations++;

	}

	// Due to the poor scheduler on some graphics card
	// we need to sort the order in which the blocks
	// are sorted to avoid poor scheduling decisions
	unsigned int sortblocks[MAXBLOCKS*2];
	for(int i = 0; i < worksize; i++)
		sortblocks[i] = ( (workset[i].end-workset[i].beg) << (int)round( log((float)(MAXBLOCKS*4.0f))/log(2.0f) ) ) + i;

	std::sort(&sortblocks[0], &sortblocks[worksize]);

	if( worksize !=0 ) {

		// Copy the block assignments to the GPU
		for(int i = 0; i < worksize; i++) {

		 	unsigned int q = ( worksize - 1 ) - ( sortblocks[i] & (MAXBLOCKS*4 - 1) );

			lqparams[i].beg = workset[q].beg;
			lqparams[i].end = workset[q].end;
			lqparams[i].flip = workset[q].flip;
			lqparams[i].sbsize = sbsize;

		}

		if( hipMemcpy(dlqparams, lqparams, worksize*sizeof(LQSortParams), hipMemcpyHostToDevice) != hipSuccess ) return;

		// Run the local quicksort, the one that doesn't need inter-block synchronization
		lqsort<<< worksize, THREADS, 2*max((THREADS+1)*2*4,sbsize*4) >>>(d_packedArrayQuick, d_auxiliaryArray, dlqparams, 0);

	}

	hipDeviceSynchronize();

	//hipFree(d_auxiliaryArray);

}

/// Clean Quick

__host__
void cleanQuick( void ) {

 	CUDA_SAFE_CALL( hipFree(d_packedArrayQuick) );

	/// Free (puqsort)
	hipHostFree( workset );
	hipHostFree( params );
	hipHostFree( length );
	hipHostFree( lqparams );
	hipFree( dparams );
	hipFree( dlqparams );
	hipFree( dhists );
	hipFree( dlength );

	hipFree(d_auxiliaryArray);

}
